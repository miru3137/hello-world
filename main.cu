#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>

/*
 * square process on device(GPU)
 */
__global__ void square(float* a, unsigned int n) {
    // get index
    unsigned int i = threadIdx.x;

    // compute square
    a[i] = a[i] * a[i];
}

/*
 * Main function
 */
int main() {
    /******************** Test Standard Library ********************/

    // print out message
    std::cout << "Hello, World!" << std::endl;

    /******************** Test CUDA Library ********************/

    // declare variables
    const unsigned int n = 20; // number of data
    float *h_a, *d_a; // host(CPU) & device(GPU) memory address

    // allocate host(CPU) memory
    h_a = new float[n];

    // allocate device(GPU) memory
    hipMalloc(&d_a, n * sizeof(float));

    // set input data
    for (unsigned int i = 0; i < n; i++)
        h_a[i] = i;

    // copy input data from host(CPU) to device(GPU)
    hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);

    // run square(^2) process
    square<<<1, n>>>(d_a, n);

    // copy output data from device(GPU) to host(CPU)
    hipMemcpy(h_a, d_a, n * sizeof(float), hipMemcpyDeviceToHost);

    // print out result
    for (unsigned int i = 0; i < n; i++)
        std::cout << h_a[i] << " ";
    std::cout << std::endl;

    /******************** Test OpenCV Library ********************/

    // read image
    const auto image = cv::imread("../lenna.png");

    // show image
    cv::imshow("Lenna", image);

    // wait key input (with update window)
    cv::waitKey();

    /******************** Exit Program ********************/

    return 0;
}
